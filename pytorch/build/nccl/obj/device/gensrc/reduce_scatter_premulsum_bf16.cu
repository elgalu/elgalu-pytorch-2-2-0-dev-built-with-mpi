#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce_scatter.h"
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_RING_LL, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_RING_LL128, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_RING_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
